#include "hip/hip_runtime.h"
#include <iostream>
#include "PublicKey.cpp"
#include <chrono>

void printResults(const int *results, unsigned long long cipher, unsigned int e,
                  unsigned int n);

/**
 * This function is run on a GPU.
 *
 * It gets the threadID (using
 * {@code unsigned long long m = blockDim.x * blockIdx.x + threadIdx.x) and
 * assumes that {@code threadID} is a possible message.
 *
 * It then calculates message ^ e modulus n iteratively and marks it as a
 * possible message by setting the value at that index to 1.
 *
 * @param cipher the cipher that is to be cracked.
 * @param e {@see PublicKey.cpp)
 * @param n {@see PublicKey.cpp)
 * @param gpu_results pointer to results. Note that this exists only in the
 * GPU memory so it must be copied back into the host.
 */
__global__
void gpu_crack(unsigned long long cipher, unsigned int e, unsigned int n,
               int *gpu_results) {
    unsigned long long possibleCipher = 1;
    unsigned long long m = blockDim.x * blockIdx.x + threadIdx.x;
    if (m < n) {
        // Instead of doing pow(cipher, e), the loop guards against overflows
        // by repeatedly taking the modulus with every multiplication.
        for (unsigned int i = 0; i < e; ++i) {
            possibleCipher *= m;
            possibleCipher = possibleCipher % n;
        }
        if (possibleCipher == cipher) {
            gpu_results[m] = 1;
            return;
        }
    }
    gpu_results[m] = 0;
}

/**
 * Parallel brute force all messages 0<=m<=n for finding a message that
 * equals to the cube root of the cipher text.
 *
 * @param cipher the cipher that is to be cracked.
 * @param publicKey the public key that is assumed to be available to the
 * hacker.
 */
void crack(unsigned long long cipher, PublicKey publicKey) {
    unsigned int e = publicKey.getE();
    unsigned int n = publicKey.getN();

    size_t size = n * sizeof(int);
    auto *results = (int *) malloc(size);
    int *gpu_results;
    hipMalloc(&gpu_results, size);


    unsigned int threadsPerBlock = 256;
    unsigned int blocksPerGrid =
            (n + threadsPerBlock - 1) / threadsPerBlock;
    gpu_crack << < blocksPerGrid, threadsPerBlock >> >
                                  (cipher, e, n, gpu_results);

    hipMemcpy(results, gpu_results, size, hipMemcpyDeviceToHost);
    printResults(results, cipher, e, n);
    hipFree(gpu_results);
    delete results;
}

void printResults(const int *results, unsigned long long cipher,
                  unsigned int e, unsigned int n) {
    bool atleast_one_result = false;
    for (unsigned int i = 0; i < n; i++) {
        if (results[i] == 1) {
            atleast_one_result = true;
            std::cout << i << "^" << e << " = " << cipher << " (mod " << n
                      << ")" << std::endl;
        }
    }
    if (!atleast_one_result) {
        std::cout << "No cube roots of " << cipher << " (mod " << n << ")";
    }
}

int main(int argc, char *argv[]) {
    unsigned long long int cipher = std::stoi(argv[1]);
    unsigned int n = std::stoi(argv[2]);
    unsigned int e = 3;
    PublicKey publicKey(n, e);
    auto start = std::chrono::high_resolution_clock::now();
    crack(cipher, publicKey);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(
            stop - start);
    std::cout << "Execution finished in " << duration.count() <<
              "ms" << std::endl;
    return 0;
}